#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>

__global__ void code_without_divergence()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a,b;
    a = b = 0;

    int wrap_id = gid / 32;

    if (wrap_id % 2 == 0 )
    {
        a = 100.0;
        b = 50.0;
    }
    else
    {
        a = 50.0;
        b = 100.0;
    }
}

__global__ void divergence_code()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a,b;
    a = b = 0;

    if (gid % 2 == 0 )
    {
        a = 100.0;
        b = 50.0;
    }
    else
    {
        a = 50.0;
        b = 100.0;
    }
}

int main()
{
    
    int size = 1 << 22;

    dim3 block_size(128);
    dim3 grid_size((size + block_size.x - 1)/ block_size.x);

    code_without_divergence << <grid_size, block_size >> > ();
    hipDeviceSynchronize();

    divergence_code << <grid_size, block_size >> > ();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}